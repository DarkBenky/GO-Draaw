#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return;}} while(0)


__global__ void vecAdd(float *x, float *y, float *z, float *x1, float *y1, float *z1, int numElements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numElements) {
        x[idx] = x[idx] + x1[idx];
        y[idx] = y[idx] + y1[idx];
        z[idx] = z[idx] + z1[idx];
    }
}

__global__ void vecSub(float *x, float *y, float *z, float *x1, float *y1, float *z1, int numElements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numElements) {
        x[idx] = x[idx] - x1[idx];
        y[idx] = y[idx] - y1[idx];
        z[idx] = z[idx] - z1[idx];
    }
}


extern "C" {

   void vectorAdd(float *x, float *y, float *z, float *x1, float *y1, float *z1, int numElements) {
        // Allocate device memory
        float *gpu_x, *gpu_y, *gpu_z, *gpu_x1, *gpu_y1, *gpu_z1;
        int memSize = numElements * sizeof(float);
        hipMalloc((void**)&gpu_x, memSize);
        hipMalloc((void**)&gpu_y, memSize);
        hipMalloc((void**)&gpu_z, memSize);
        hipMalloc((void**)&gpu_x1, memSize);
        hipMalloc((void**)&gpu_y1, memSize);
        hipMalloc((void**)&gpu_z1, memSize);

        // Copy input vectors from host memory to device memory
        hipMemcpy(gpu_x, x, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_y, y, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_z, z, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_x1, x1, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_y1, y1, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_z1, z1, memSize, hipMemcpyHostToDevice);

        // Calculate grid and block dimensions
        int threadsPerBlock = 32;
        int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

        // Launch the vector addition kernel
        vecAdd<<<blocksPerGrid, threadsPerBlock>>>(gpu_x, gpu_y, gpu_z, gpu_x1, gpu_y1, gpu_z1, numElements);

        // Check for kernel launch errors
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        }

        // Wait for kernel to finish
        hipDeviceSynchronize();

        // Copy result from device memory to host memory
        hipMemcpy(x, gpu_x, memSize, hipMemcpyDeviceToHost);
        hipMemcpy(y, gpu_y, memSize, hipMemcpyDeviceToHost);
        hipMemcpy(z, gpu_z, memSize, hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(gpu_x);
        hipFree(gpu_y);
        hipFree(gpu_z);
        hipFree(gpu_x1);
        hipFree(gpu_y1);
        hipFree(gpu_z1);
    }

     void vectorSub(float *x, float *y, float *z, float *x1, float *y1, float *z1, int numElements) {
        // Allocate device memory
        float *gpu_x, *gpu_y, *gpu_z, *gpu_x1, *gpu_y1, *gpu_z1;
        int memSize = numElements * sizeof(float);
        hipMalloc((void**)&gpu_x, memSize);
        hipMalloc((void**)&gpu_y, memSize);
        hipMalloc((void**)&gpu_z, memSize);
        hipMalloc((void**)&gpu_x1, memSize);
        hipMalloc((void**)&gpu_y1, memSize);
        hipMalloc((void**)&gpu_z1, memSize);

        // Copy input vectors from host memory to device memory
        hipMemcpy(gpu_x, x, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_y, y, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_z, z, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_x1, x1, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_y1, y1, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_z1, z1, memSize, hipMemcpyHostToDevice);

        // Calculate grid and block dimensions
        int threadsPerBlock = 32;
        int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

        // Launch the vector subtraction kernel
        vecSub<<<blocksPerGrid, threadsPerBlock>>>(gpu_x, gpu_y, gpu_z, gpu_x1, gpu_y1, gpu_z1, numElements);

        // Check for kernel launch errors
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        }

        // Wait for kernel to finish
        hipDeviceSynchronize();

        // Copy result from device memory to host memory
        hipMemcpy(x, gpu_x, memSize, hipMemcpyDeviceToHost);
        hipMemcpy(y, gpu_y, memSize, hipMemcpyDeviceToHost);
        hipMemcpy(z, gpu_z, memSize, hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(gpu_x);
        hipFree(gpu_y);
        hipFree(gpu_z);
        hipFree(gpu_x1);
        hipFree(gpu_y1);
        hipFree(gpu_z1);
    }
}
