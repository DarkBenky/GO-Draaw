#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return;}} while(0)


__global__ void vecAdd(float *x, float *y, float *z, float *x1, float *y1, float *z1, int numElements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numElements) {
        x[idx] = x[idx] + x1[idx];
        y[idx] = y[idx] + y1[idx];
        z[idx] = z[idx] + z1[idx];
    }
}

__global__ void vecSub(float *x, float *y, float *z, float *x1, float *y1, float *z1, int numElements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numElements) {
        x[idx] = x[idx] - x1[idx];
        y[idx] = y[idx] - y1[idx];
        z[idx] = z[idx] - z1[idx];
    }
}

__global__ void Normalize(float *x, float *y, float *z, int numVectors) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numVectors) {
        float magnitude = sqrt(x[idx] * x[idx] + y[idx] * y[idx] + z[idx] * z[idx]);
        if (magnitude > 0.0f) {
            x[idx] /= magnitude;
            y[idx] /= magnitude;
            z[idx] /= magnitude;
        }
    }
}

__global__ void vecDot(float *x, float *y, float *z, float *x1, float *y1, float *z1, float *dotProduct, int numElements) {
    __shared__ float cache[256];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int cacheIdx = threadIdx.x;

    float temp = 0;
    if (idx < numElements) {
        temp = x[idx] * x1[idx] + y[idx] * y1[idx] + z[idx] * z1[idx];
    }

    cache[cacheIdx] = temp;

    __syncthreads();

    // Reduction
    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIdx < i) {
            cache[cacheIdx] += cache[cacheIdx + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIdx == 0) {
        atomicAdd(dotProduct, cache[0]);
    }
}

__global__ void vecCross(float *x, float *y, float *z, float *x1, float *y1, float *z1, float *cx, float *cy, float *cz, int numElements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numElements) {
        cx[idx] = y[idx] * z1[idx] - z[idx] * y1[idx];
        cy[idx] = z[idx] * x1[idx] - x[idx] * z1[idx];
        cz[idx] = x[idx] * y1[idx] - y[idx] * x1[idx];
    }
}


extern "C" {

    void vectorNormalize(float *x, float *y, float *z, int numElements) {
        float *gpu_x, *gpu_y, *gpu_z;
        int memSize = numElements * sizeof(float);

        hipMalloc((void**)&gpu_x, memSize);
        hipMalloc((void**)&gpu_y, memSize);
        hipMalloc((void**)&gpu_z, memSize);

        hipMemcpy(gpu_x, x, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_y, y, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_z, z, memSize, hipMemcpyHostToDevice);

        int threadsPerBlock = 256;
        int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

        Normalize<<<blocksPerGrid, threadsPerBlock>>>(gpu_x, gpu_y, gpu_z, numElements);
        hipDeviceSynchronize();

        hipMemcpy(x, gpu_x, memSize, hipMemcpyDeviceToHost);
        hipMemcpy(y, gpu_y, memSize, hipMemcpyDeviceToHost);
        hipMemcpy(z, gpu_z, memSize, hipMemcpyDeviceToHost);

        hipFree(gpu_x);
        hipFree(gpu_y);
        hipFree(gpu_z);
    }

    void vectorDot(float *x, float *y, float *z, float *x1, float *y1, float *z1, float *result, int numElements) {
        float *gpu_x, *gpu_y, *gpu_z, *gpu_x1, *gpu_y1, *gpu_z1, *gpu_result;
        int memSize = numElements * sizeof(float);
        hipMalloc((void**)&gpu_x, memSize);
        hipMalloc((void**)&gpu_y, memSize);
        hipMalloc((void**)&gpu_z, memSize);
        hipMalloc((void**)&gpu_x1, memSize);
        hipMalloc((void**)&gpu_y1, memSize);
        hipMalloc((void**)&gpu_z1, memSize);
        hipMalloc((void**)&gpu_result, sizeof(float));

        hipMemcpy(gpu_x, x, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_y, y, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_z, z, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_x1, x1, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_y1, y1, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_z1, z1, memSize, hipMemcpyHostToDevice);

        hipMemset(gpu_result, 0, sizeof(float));

        int threadsPerBlock = 256;
        int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

        vecDot<<<blocksPerGrid, threadsPerBlock>>>(gpu_x, gpu_y, gpu_z, gpu_x1, gpu_y1, gpu_z1, gpu_result, numElements);
        hipDeviceSynchronize();

        hipMemcpy(result, gpu_result, sizeof(float), hipMemcpyDeviceToHost);

        hipFree(gpu_x);
        hipFree(gpu_y);
        hipFree(gpu_z);
        hipFree(gpu_x1);
        hipFree(gpu_y1);
        hipFree(gpu_z1);
        hipFree(gpu_result);
    }

    void vectorCross(float *x, float *y, float *z, float *x1, float *y1, float *z1, float *cx, float *cy, float *cz, int numElements) {
        float *gpu_x, *gpu_y, *gpu_z, *gpu_x1, *gpu_y1, *gpu_z1, *gpu_cx, *gpu_cy, *gpu_cz;
        int memSize = numElements * sizeof(float);
        hipMalloc((void**)&gpu_x, memSize);
        hipMalloc((void**)&gpu_y, memSize);
        hipMalloc((void**)&gpu_z, memSize);
        hipMalloc((void**)&gpu_x1, memSize);
        hipMalloc((void**)&gpu_y1, memSize);
        hipMalloc((void**)&gpu_z1, memSize);
        hipMalloc((void**)&gpu_cx, memSize);
        hipMalloc((void**)&gpu_cy, memSize);
        hipMalloc((void**)&gpu_cz, memSize);

        hipMemcpy(gpu_x, x, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_y, y, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_z, z, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_x1, x1, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_y1, y1, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_z1, z1, memSize, hipMemcpyHostToDevice);

        int threadsPerBlock = 256;
        int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

        vecCross<<<blocksPerGrid, threadsPerBlock>>>(gpu_x, gpu_y, gpu_z, gpu_x1, gpu_y1, gpu_z1, gpu_cx, gpu_cy, gpu_cz, numElements);
        hipDeviceSynchronize();

        hipMemcpy(cx, gpu_cx, memSize, hipMemcpyDeviceToHost);
        hipMemcpy(cy, gpu_cy, memSize, hipMemcpyDeviceToHost);
        hipMemcpy(cz, gpu_cz, memSize, hipMemcpyDeviceToHost);

        hipFree(gpu_x);
        hipFree(gpu_y);
        hipFree(gpu_z);
        hipFree(gpu_x1);
        hipFree(gpu_y1);
        hipFree(gpu_z1);
        hipFree(gpu_cx);
        hipFree(gpu_cy);
        hipFree(gpu_cz);
    }

   void vectorAdd(float *x, float *y, float *z, float *x1, float *y1, float *z1, int numElements) {
        // Allocate device memory
        float *gpu_x, *gpu_y, *gpu_z, *gpu_x1, *gpu_y1, *gpu_z1;
        int memSize = numElements * sizeof(float);
        hipMalloc((void**)&gpu_x, memSize);
        hipMalloc((void**)&gpu_y, memSize);
        hipMalloc((void**)&gpu_z, memSize);
        hipMalloc((void**)&gpu_x1, memSize);
        hipMalloc((void**)&gpu_y1, memSize);
        hipMalloc((void**)&gpu_z1, memSize);

        // Copy input vectors from host memory to device memory
        hipMemcpy(gpu_x, x, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_y, y, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_z, z, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_x1, x1, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_y1, y1, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_z1, z1, memSize, hipMemcpyHostToDevice);

        // Calculate grid and block dimensions
        int threadsPerBlock = 32;
        int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

        // Launch the vector addition kernel
        vecAdd<<<blocksPerGrid, threadsPerBlock>>>(gpu_x, gpu_y, gpu_z, gpu_x1, gpu_y1, gpu_z1, numElements);

        // Check for kernel launch errors
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        }

        // Wait for kernel to finish
        hipDeviceSynchronize();

        // Copy result from device memory to host memory
        hipMemcpy(x, gpu_x, memSize, hipMemcpyDeviceToHost);
        hipMemcpy(y, gpu_y, memSize, hipMemcpyDeviceToHost);
        hipMemcpy(z, gpu_z, memSize, hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(gpu_x);
        hipFree(gpu_y);
        hipFree(gpu_z);
        hipFree(gpu_x1);
        hipFree(gpu_y1);
        hipFree(gpu_z1);
    }

     void vectorSub(float *x, float *y, float *z, float *x1, float *y1, float *z1, int numElements) {
        // Allocate device memory
        float *gpu_x, *gpu_y, *gpu_z, *gpu_x1, *gpu_y1, *gpu_z1;
        int memSize = numElements * sizeof(float);
        hipMalloc((void**)&gpu_x, memSize);
        hipMalloc((void**)&gpu_y, memSize);
        hipMalloc((void**)&gpu_z, memSize);
        hipMalloc((void**)&gpu_x1, memSize);
        hipMalloc((void**)&gpu_y1, memSize);
        hipMalloc((void**)&gpu_z1, memSize);

        // Copy input vectors from host memory to device memory
        hipMemcpy(gpu_x, x, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_y, y, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_z, z, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_x1, x1, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_y1, y1, memSize, hipMemcpyHostToDevice);
        hipMemcpy(gpu_z1, z1, memSize, hipMemcpyHostToDevice);

        // Calculate grid and block dimensions
        int threadsPerBlock = 32;
        int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

        // Launch the vector subtraction kernel
        vecSub<<<blocksPerGrid, threadsPerBlock>>>(gpu_x, gpu_y, gpu_z, gpu_x1, gpu_y1, gpu_z1, numElements);

        // Check for kernel launch errors
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        }

        // Wait for kernel to finish
        hipDeviceSynchronize();

        // Copy result from device memory to host memory
        hipMemcpy(x, gpu_x, memSize, hipMemcpyDeviceToHost);
        hipMemcpy(y, gpu_y, memSize, hipMemcpyDeviceToHost);
        hipMemcpy(z, gpu_z, memSize, hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(gpu_x);
        hipFree(gpu_y);
        hipFree(gpu_z);
        hipFree(gpu_x1);
        hipFree(gpu_y1);
        hipFree(gpu_z1);
    }
}